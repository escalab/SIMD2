#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <cublas.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <sys/time.h>
#include <time.h>

// #include "../kernel/mmo_kernel.cuh"

#define BLOCK_DIM 16


/**
 * Computes the squared Euclidean distance matrix between the query points and the reference points.
 *
 * @param ref          refence points stored in the global memory
 * @param ref_width    number of reference points
 * @param ref_pitch    pitch of the reference points array in number of column
 * @param query        query points stored in the global memory
 * @param query_width  number of query points
 * @param query_pitch  pitch of the query points array in number of columns
 * @param height       dimension of points = height of texture `ref` and of the array `query`
 * @param dist         array containing the query_width x ref_width computed distances
 */
__global__ void compute_distances(float * ref,
                                  int     ref_width,
                                  int     ref_pitch,
                                  float * query,
                                  int     query_width,
                                  int     query_pitch,
                                  int     height,
                                  float * dist) {

    // Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
    __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

    // Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
    __shared__ int begin_A;
    __shared__ int begin_B;
    __shared__ int step_A;
    __shared__ int step_B;
    __shared__ int end_A;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Initializarion of the SSD for the current thread
    float ssd = 0.f;

    // Loop parameters
    begin_A = BLOCK_DIM * blockIdx.y;
    begin_B = BLOCK_DIM * blockIdx.x;
    step_A  = BLOCK_DIM * ref_pitch;
    step_B  = BLOCK_DIM * query_pitch;
    end_A   = begin_A + (height-1) * ref_pitch;

    // Conditions
    int cond0 = (begin_A + tx < ref_width); // used to write in shared memory
    int cond1 = (begin_B + tx < query_width); // used to write in shared memory & to computations and to write in output array 
    int cond2 = (begin_A + ty < ref_width); // used to computations and to write in output matrix

    // Loop over all the sub-matrices of A and B required to compute the block sub-matrix
    for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {

        // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
        if (a/ref_pitch + ty < height) {
            shared_A[ty][tx] = (cond0)? ref[a + ref_pitch * ty + tx] : 0;
            shared_B[ty][tx] = (cond1)? query[b + query_pitch * ty + tx] : 0;
        }
        else {
            shared_A[ty][tx] = 0;
            shared_B[ty][tx] = 0;
        }

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
        if (cond2 && cond1) {
            for (int k = 0; k < BLOCK_DIM; ++k){
                float tmp = shared_A[k][ty] - shared_B[k][tx];
                ssd += tmp*tmp;
            }
        }

        // Synchronize to make sure that the preceeding computation is done before loading two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory; each thread writes one element
    if (cond2 && cond1) {
        dist[ (begin_A + ty) * query_pitch + begin_B + tx ] = ssd;
    }
}



/**
 * For each reference point (i.e. each column) finds the k-th smallest distances
 * of the distance matrix and their respective indexes and gathers them at the top
 * of the 2 arrays.
 *
 * Since we only need to locate the k smallest distances, sorting the entire array
 * would not be very efficient if k is relatively small. Instead, we perform a
 * simple insertion sort by eventually inserting a given distance in the first
 * k values.
 *
 * @param dist         distance matrix
 * @param dist_pitch   pitch of the distance matrix given in number of columns
 * @param index        index matrix
 * @param index_pitch  pitch of the index matrix given in number of columns
 * @param width        width of the distance matrix and of the index matrix
 * @param height       height of the distance matrix
 * @param k            number of values to find
 */
__global__ void modified_insertion_sort(float * dist,
                                        int     dist_pitch,
                                        int *   index,
                                        int     index_pitch,
                                        int     width,
                                        int     height,
                                        int     k){

    // Column position
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Do nothing if we are out of bounds
    if (xIndex < width) {

        // Pointer shift
        float * p_dist  = dist  + xIndex;
        int *   p_index = index + xIndex;

        // Initialise the first index
        p_index[0] = 0;

        // Go through all points
        for (int i=1; i<height; ++i) {

            // Store current distance and associated index
            float curr_dist = p_dist[i*dist_pitch];
            int   curr_index  = i;

            // Skip the current value if its index is >= k and if it's higher the k-th slready sorted mallest value
            if (i >= k && curr_dist >= p_dist[(k-1)*dist_pitch]) {
                continue;
            }

            // Shift values (and indexes) higher that the current distance to the right
            int j = min(i, k-1);
            while (j > 0 && p_dist[(j-1)*dist_pitch] > curr_dist) {
                p_dist[j*dist_pitch]   = p_dist[(j-1)*dist_pitch];
                p_index[j*index_pitch] = p_index[(j-1)*index_pitch];
                --j;
            }

            // Write the current distance and index at their position
            p_dist[j*dist_pitch]   = curr_dist;
            p_index[j*index_pitch] = curr_index; 
        }
    }
}

/**
 * Computes the square root of the first k lines of the distance matrix.
 *
 * @param dist   distance matrix
 * @param width  width of the distance matrix
 * @param pitch  pitch of the distance matrix given in number of columns
 * @param k      number of values to consider
 */
__global__ void compute_sqrt(float * dist, int width, int pitch, int k){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex<width && yIndex<k)
        dist[yIndex*pitch + xIndex] = sqrt(dist[yIndex*pitch + xIndex]);
}




bool knn_cuda_global(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,
                     int *         knn_index,
                     double *       kernel_time) {

    // Constants
    const unsigned int size_of_float = sizeof(float);
    const unsigned int size_of_int   = sizeof(int);

    struct timeval tic;
   

    // Return variables
    hipError_t err0, err1, err2, err3;

    // Check that we have at least one CUDA device 
    int nb_devices;
    err0 = hipGetDeviceCount(&nb_devices);
    if (err0 != hipSuccess || nb_devices == 0) {
        printf("ERROR: No CUDA device found\n");
        return false;
    }

    // Select the first CUDA device as default
    err0 = hipSetDevice(0);
    if (err0 != hipSuccess) {
        printf("ERROR: Cannot set the chosen CUDA device\n");
        return false;
    }

    // Allocate global memory
    float * ref_dev   = NULL;
    float * query_dev = NULL;
    float * dist_dev  = NULL;
    int   * index_dev = NULL;
    size_t  ref_pitch_in_bytes;
    size_t  query_pitch_in_bytes;
    size_t  dist_pitch_in_bytes;
    size_t  index_pitch_in_bytes;
    err0 = hipMallocPitch((void**)&ref_dev,   &ref_pitch_in_bytes,   ref_nb   * size_of_float, dim);
    err1 = hipMallocPitch((void**)&query_dev, &query_pitch_in_bytes, query_nb * size_of_float, dim);
    err2 = hipMallocPitch((void**)&dist_dev,  &dist_pitch_in_bytes,  query_nb * size_of_float, ref_nb);
    err3 = hipMallocPitch((void**)&index_dev, &index_pitch_in_bytes, query_nb * size_of_int,   k);
    if (err0 != hipSuccess || err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess) {
        printf("ERROR: Memory allocation error\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Deduce pitch values
    size_t ref_pitch   = ref_pitch_in_bytes   / size_of_float;
    size_t query_pitch = query_pitch_in_bytes / size_of_float;
    size_t dist_pitch  = dist_pitch_in_bytes  / size_of_float;
    size_t index_pitch = index_pitch_in_bytes / size_of_int;

    // Check pitch values
    if (query_pitch != dist_pitch || query_pitch != index_pitch) {
        printf("ERROR: Invalid pitch value\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    // Copy reference and query data from the host to the device
    err0 = hipMemcpy2D(ref_dev,   ref_pitch_in_bytes,   ref,   ref_nb * size_of_float,   ref_nb * size_of_float,   dim, hipMemcpyHostToDevice);
    err1 = hipMemcpy2D(query_dev, query_pitch_in_bytes, query, query_nb * size_of_float, query_nb * size_of_float, dim, hipMemcpyHostToDevice);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from host to device\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    gettimeofday(&tic, NULL);
    // Compute the squared Euclidean distances
    dim3 block0(BLOCK_DIM, BLOCK_DIM, 1);
    dim3 grid0(query_nb / BLOCK_DIM, ref_nb / BLOCK_DIM, 1);
    if (query_nb % BLOCK_DIM != 0) grid0.x += 1;
    if (ref_nb   % BLOCK_DIM != 0) grid0.y += 1;
    compute_distances<<<grid0, block0>>>(ref_dev, ref_nb, ref_pitch, query_dev, query_nb, query_pitch, dim, dist_dev);
    
    hipDeviceSynchronize();
    

    
    struct timeval toc;
    gettimeofday(&toc, NULL);
    double elapsed_time = toc.tv_sec - tic.tv_sec;
    *kernel_time += elapsed_time + (toc.tv_usec - tic.tv_usec) / 1000000.;
    // printf(" in %.0f us (averaged over %3d iterations)\n", (elapsed_time / nb_iterations)*1000000, nb_iterations);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }
    // Sort the distances with their respective indexes
    dim3 block1(256, 1, 1);
    dim3 grid1(query_nb / 256, 1, 1);
    if (query_nb % 256 != 0) grid1.x += 1;
    modified_insertion_sort<<<grid1, block1>>>(dist_dev, dist_pitch, index_dev, index_pitch, query_nb, ref_nb, k);
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Compute the square root of the k smallest distances
    dim3 block2(16, 16, 1);
    dim3 grid2(query_nb / 16, k / 16, 1);
    if (query_nb % 16 != 0) grid2.x += 1;
    if (k % 16 != 0)        grid2.y += 1;
    compute_sqrt<<<grid2, block2>>>(dist_dev, query_nb, query_pitch, k);	
    if (hipGetLastError() != hipSuccess) {
        printf("ERROR: Unable to execute kernel\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false;
    }

    // Copy k smallest distances / indexes from the device to the host
    err0 = hipMemcpy2D(knn_dist,  query_nb * size_of_float, dist_dev,  dist_pitch_in_bytes,  query_nb * size_of_float, k, hipMemcpyDeviceToHost);
    err1 = hipMemcpy2D(knn_index, query_nb * size_of_int,   index_dev, index_pitch_in_bytes, query_nb * size_of_int,   k, hipMemcpyDeviceToHost);
    if (err0 != hipSuccess || err1 != hipSuccess) {
        printf("ERROR: Unable to copy data from device to host\n");
        hipFree(ref_dev);
        hipFree(query_dev);
        hipFree(dist_dev);
        hipFree(index_dev); 
        return false; 
    }

    // Memory clean-up
    hipFree(ref_dev);
    hipFree(query_dev);
    hipFree(dist_dev);
    hipFree(index_dev); 

    return true;
}