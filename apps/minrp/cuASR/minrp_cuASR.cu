#include "../../../kernel/srgemm.cuh"
#include "../../../kernel/precision.cuh"
#include "../../../kernel/converge.cuh"
#include "../../../utils/check_sum.h"
#include "../../../utils/print_mat.h"
#include "../../data/graph_gen.h"
#include <sys/time.h>

#include <float.h>
#include <chrono>
#include <iostream>
#include <unistd.h>

#define NUM_ITR 10



double minrp_kernel(float * adj_mat, float * dist, int v) {
    using namespace std::chrono;

    float * adj_mat_d; // original graph adj matrix
    float * out_d_delta; // execution result of previous run.
    float * out_d;    // new dist matrix after latest execution

    int * check_d;
    int * check_h;

    check_h = (int*)malloc(sizeof(int));
    hipMalloc((int**)&check_d, sizeof(int));
  
    hipMalloc((float**)&adj_mat_d,v*v*sizeof(float));
    hipMalloc((float**)&out_d, v*v*sizeof(float));
    hipMalloc((float**)&out_d_delta, v*v*sizeof(float));
  
    //move data (same value initially)
    hipMemcpy(adj_mat_d, adj_mat, v*v*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_d_delta, adj_mat, v*v*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_d, adj_mat, v*v*sizeof(float), hipMemcpyHostToDevice);

    bool run = true;
    int num_itr = 0;
    auto start  = high_resolution_clock::now();

    // maximum of 500 iterations of srgemm, wont affect graph with diameter < 500
    while(run && (num_itr < 500)){ 
        num_itr += 1;
        // 1 iteration of minplus srgemm
        int retval = cuasr_minmul_srsgemm(v, v, v, \
                                        out_d, v, \
                                        out_d, v, \
                                        out_d, v, \
                                        out_d_delta, \
                                        true, nullptr);
        hipDeviceSynchronize();
        // check convergence
        run = comp_update(out_d, out_d_delta, check_d, check_h, v,v);
    }

    hipDeviceSynchronize();
    auto end    = high_resolution_clock::now();
    auto delta = duration_cast<nanoseconds>(end - start).count();
    double rt = (double)delta / 1000000;
    hipMemcpy(dist, out_d, v*v*sizeof(float), hipMemcpyDeviceToHost);

    // printf("num_itr: %d\n", num_itr);
    hipFree(adj_mat_d);
    hipFree(out_d_delta);
    hipFree(out_d);
    hipFree(check_d);
    free(check_h);
    return rt;
}

int main(int argc, char *argv[]){
    int v;
    int e;
    // int bound;
    // float edge_weight = 10;
    // float density = 0.01;
    // int i,j; // looper
    float *adj_mat; // init adj_mat
    
    // if (strcmp(argv[1], "-g")){
    int v1, v2;
    float value;
    std::cin >> v >> e;
    adj_mat = (float*)malloc(v * v * sizeof(float));
    for (int i = 0; i < v*v; i++){
        adj_mat[i] = FLT_MAX;
    }
    for (int i=0; i < e; ++i) {
        std::cin >> v1 >> v2 >> value;
        adj_mat[v1 * v + v2] = value;
    }

    float * dist_tensor;
    dist_tensor = (float*)calloc(v * v, sizeof(float));
    if (!dist_tensor){
        printf("failed to malloc dist_tensor\n");
    }
    double rt;
    for(int i = 0; i < NUM_ITR; i++){
        rt += minrp_kernel(adj_mat,dist_tensor,v);
    }
    
    // float cs = check_sum<float>(dist_tensor, v*v);
    
    // print_matrix<float>(dist_tensor,v,v);

    free(adj_mat);
    free(dist_tensor);
    printf("%f\n",rt/NUM_ITR);
    // printf("apsp_cuASR,    check-sum: %f\n",cs);
    return 0;
}