/*
 * =======================================================================
 *  This file is part of APSP-CUDA.
 *  Copyright (C) 2016 Marios Mitalidis
 *
 *  APSP-CUDA is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  APSP-CUDA is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with APSP-CUDA.  If not, see <http://www.gnu.org/licenses/>.
 * =======================================================================
 */ 


#include "cuda_error_check.h"
#include <cstdio>

void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void gpuKerAssert(const char* file, int line)
{
	gpuAssert( hipPeekAtLastError(), file, line );
	gpuAssert( hipDeviceSynchronize(), file, line );
}
