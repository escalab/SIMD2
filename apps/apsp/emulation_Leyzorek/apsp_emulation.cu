#include "hip/hip_runtime.h"
#include "../../../kernel/srgemm.cuh"
#include "../../../kernel/tensor_srgemm.cuh"
#include "../../../kernel/precision.cuh"
#include "../../../kernel/converge.cuh"
#include "../../../utils/check_sum.h"
#include "../../data/graph_gen.h"

#include <hip/hip_runtime_api.h>

#include <sys/time.h>

#include <float.h>
#include <chrono>

#define NUM_ITR 20
#define PERFORM

double apsp_kernel(float * adj_mat, float * dist_tensor, int v, int num_itrs, hipblasHandle_t cublasHandle){
    using namespace std::chrono;
    float * adj_mat_d; // original graph adj matrix
    float * out_d_delta; // execution result of previous run.
    float * out_d;    // new dist matrix after latest execution

    half * adj_mat_d_fp16; // original graph adj matrix
    half * out_d_delta_fp16; // execution result of previous run.

    int * check_d;
    int * check_h;

    check_h = (int*)malloc(sizeof(int));
    hipMalloc((int**)&check_d, sizeof(int));
  
    hipMalloc((float**)&adj_mat_d,v*v*sizeof(float));
    hipMalloc((float**)&out_d, v*v*sizeof(float));
    hipMalloc((float**)&out_d_delta, v*v*sizeof(float));

    hipMalloc((half**)&out_d_delta_fp16, v*v*sizeof(half));
    hipMalloc((half**)&adj_mat_d_fp16,v*v*sizeof(half));

    hipMemcpy(adj_mat_d, adj_mat, v*v*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_d_delta, adj_mat, v*v*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_d, adj_mat, v*v*sizeof(float), hipMemcpyHostToDevice);

    // conversion
    f2h_device(adj_mat_d_fp16, adj_mat_d, v*v);
    f2h_device(out_d_delta_fp16, out_d_delta, v*v);

    auto start  = high_resolution_clock::now();
    for(int i = 0; i < num_itrs; i ++){
        cublas_gemmEx(adj_mat_d_fp16, out_d_delta_fp16, out_d, out_d, v, v, v, 1.0, 1,cublasHandle);
        f2h_device(out_d_delta_fp16, out_d, v*v);
    }
    hipDeviceSynchronize();
    auto end    = high_resolution_clock::now();
    auto delta = duration_cast<nanoseconds>(end - start).count();
    double rt = (double)delta / 1000000;

    hipFree(adj_mat_d);
    hipFree(out_d_delta);
    hipFree(out_d);
    hipFree(adj_mat_d_fp16);
    hipFree(out_d_delta_fp16);
    hipFree(check_d);
    free(check_h);
    return rt;

}


int apsp_itr_leyz(float * adj_mat, float * dist, int v) {

    float * adj_mat_d; // original graph adj matrix
    float * out_d_delta; // execution result of previous run.
    float * out_d;    // new dist matrix after latest execution

    int * check_d;
    int * check_h;

    check_h = (int*)malloc(sizeof(int));
    hipMalloc((int**)&check_d, sizeof(int));
  
    hipMalloc((float**)&adj_mat_d,v*v*sizeof(float));
    hipMalloc((float**)&out_d, v*v*sizeof(float));
    hipMalloc((float**)&out_d_delta, v*v*sizeof(float));
  
    //move data (same value initially)
    hipMemcpy(adj_mat_d, adj_mat, v*v*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_d_delta, adj_mat, v*v*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_d, adj_mat, v*v*sizeof(float), hipMemcpyHostToDevice);

    bool run = true;
    int num_itr = 0;
    // maximum of 500 iterations of srgemm, wont affect graph with diameter < 500
    while(run && (num_itr < 500)){ 
        num_itr += 1;
        // 1 iteration of minplus srgemm
        int retval = cuasr_minplus_srsgemm(v, v, v, \
                                        out_d, v, \
                                        out_d, v, \
                                        out_d, v, \
                                        out_d_delta, \
                                        true, nullptr);
        hipDeviceSynchronize();
        // check convergence
        run = comp_update(out_d, out_d_delta, check_d, check_h, v,v);
    }
    hipMemcpy(dist, out_d, v*v*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(adj_mat_d);
    hipFree(out_d_delta);
    hipFree(out_d);
    hipFree(check_d);
    free(check_h);
    return num_itr;
}

int apsp_itr(float * adj_mat, float * dist, int v) {

    float * adj_mat_d; // original graph adj matrix
    float * out_d_delta; // execution result of previous run.
    float * out_d;    // new dist matrix after latest execution

    int * check_d;
    int * check_h;

    check_h = (int*)malloc(sizeof(int));
    hipMalloc((int**)&check_d, sizeof(int));
  
    hipMalloc((float**)&adj_mat_d,v*v*sizeof(float));
    hipMalloc((float**)&out_d, v*v*sizeof(float));
    hipMalloc((float**)&out_d_delta, v*v*sizeof(float));
  
    //move data (same value initially)
    hipMemcpy(adj_mat_d, adj_mat, v*v*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_d_delta, adj_mat, v*v*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_d, adj_mat, v*v*sizeof(float), hipMemcpyHostToDevice);

    bool run = true;
    int num_itr = 0;
    // maximum of 500 iterations of srgemm, wont affect graph with diameter < 500
    while(run && (num_itr < v)){ 
        num_itr += 1;
        // 1 iteration of minplus srgemm
        int retval = cuasr_minplus_srsgemm(v, v, v, \
                                        adj_mat_d, v, \
                                        out_d, v, \
                                        out_d, v, \
                                        out_d_delta, \
                                        true, nullptr);
        hipDeviceSynchronize();
        // check convergence
        run = comp_update(out_d, out_d_delta, check_d, check_h, v,v);
    }
    hipMemcpy(dist, out_d, v*v*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(adj_mat_d);
    hipFree(out_d_delta);
    hipFree(out_d);
    hipFree(check_d);
    free(check_h);
    return num_itr;
}

int main(int argc, char *argv[]){
    int v;
    int e;
    // int bound;
    float edge_weight;
    float density;
    // int i,j; // looper
    float *adj_mat; // init adj_mat
   
    if (!strcmp(argv[1], "-f")){
        int v1, v2;// value;
        float value;
        std::cin >> v >> e;
        adj_mat = (float*)malloc(v * v * sizeof(float));
        for (int i = 0; i < v*v; i++){
            adj_mat[i] = FLT_MAX;
        }
        for (int i=0; i < e; ++i) {
            std::cin >> v1 >> v2 >> value;
            adj_mat[v1 * v + v2] = -(float)value;
        }
        for(int i = 0; i < v; i++){
            adj_mat[i*v+i] = 0;
        }
        // add I/O
    }
    else{
        if (argc < 4){
            printf("Usage: ./apsp-cuda-v3 num_vertices density edge_weight\n");
            printf("    number of edges = num_vertices * density\n");
            printf("    max edge weight = edge_weight\n");
            exit(0);
        }
        v = atoi(argv[1]);
        density = atof(argv[2]);
        if (density < 0 || density > 1){
            printf("Input density %.2f not within range 0 - 1\n",density);
            exit(0);
        }
        edge_weight = atof(argv[3]);
        // bound = atoi(argv[4]);
        adj_mat = (float*)malloc(v * v * sizeof(float));
        e = rgg_1d(adj_mat, v, density, edge_weight, 7);
    }
  
    
    float * dist_tensor;
    dist_tensor = (float*)calloc(v * v, sizeof(float));
    if (!dist_tensor){
        printf("failed to malloc dist_tensor\n");
        printf("v = %d\n", v);
    }


    int num_itrs = apsp_itr(adj_mat,dist_tensor,v);
    float cs = check_sum<float>(dist_tensor, v*v);
    printf("apsp_cuASR_old,    check-sum: %f\n",cs);
    printf("iters: %d\n",num_itrs);

    num_itrs = apsp_itr_leyz(adj_mat,dist_tensor,v);
    cs = check_sum<float>(dist_tensor, v*v);
    printf("apsp_cuASR_leyz,    check-sum: %f\n",cs);
    printf("iters: %d\n",num_itrs);

    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    apsp_kernel(adj_mat,dist_tensor,v, num_itrs,cublasHandle);


    double rt = 0.0;
    #ifdef PERFORM
    // for (int i = 0 ; i <  NUM_ITR; i++){
    //     rt += apsp_kernel(adj_mat,dist_tensor,v, num_itrs,cublasHandle);
    // }
    #endif
    
    

    hipblasDestroy(cublasHandle);
    free(adj_mat);
    free(dist_tensor);
    // printf("%f %d\n",rt/(double)NUM_ITR, num_itrs);
    // printf("%d\n", num_itrs);
    
    return 0;
}