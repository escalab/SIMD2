#include "fwgpu/utils.hpp"

#include <hip/hip_runtime.h>

namespace fwgpu {

auto malloc_device(void **dptr, size_t size) -> int {
  auto retval = static_cast<int>(hipMalloc(dptr, size));
  return retval;
}

auto malloc_unified(void **dptr, size_t size) -> int {
  auto retval = static_cast<int>(hipMallocManaged(dptr, size));
  return retval;
}

auto memcpy_inferred(void *dest, const void *src, size_t size) -> int {
  auto retval = static_cast<int>(hipMemcpy(dest, src, size, hipMemcpyDefault));
  return retval;
}

auto free_device(void *dbuf) -> int {
  auto retval = static_cast<int>(hipFree(dbuf));
  return retval;
}

auto memcpy_d2h(void *dest, const void *src, size_t size) -> int {
  auto retval = static_cast<int>(hipMemcpy(dest, src, size, hipMemcpyDeviceToHost));
  return retval;
}

auto memcpy_h2d(void *dest, const void *src, size_t size) -> int {
  auto retval = static_cast<int>(hipMemcpy(dest, src, size, hipMemcpyHostToDevice));
  return retval;
}

auto memcpy_h2h(void *dest, const void *src, size_t size) -> int {
  auto retval = static_cast<int>(hipMemcpy(dest, src, size, hipMemcpyDeviceToDevice));
  return retval;
}

auto memcpy_d2d(void *dest, const void *src, size_t size) -> int {
  auto retval = static_cast<int>(hipMemcpy(dest, src, size, hipMemcpyHostToHost));
  return retval;
}

auto memcpy_2d_h2d(
    void *dest,
    size_t dpitch,
    const void *src,
    size_t spitch,
    size_t width,
    size_t height) -> int {
  auto retval = static_cast<int>(
      hipMemcpy2D(dest, dpitch, src, spitch, width, height, hipMemcpyHostToDevice));
  return retval;
}

auto memcpy_2d_d2h(
    void *dest,
    size_t dpitch,
    const void *src,
    size_t spitch,
    size_t width,
    size_t height) -> int {
  auto retval = static_cast<int>(
      hipMemcpy2D(dest, dpitch, src, spitch, width, height, hipMemcpyDeviceToHost));
  return retval;
}

auto memcpy_2d_d2d(
    void *dest,
    size_t dpitch,
    const void *src,
    size_t spitch,
    size_t width,
    size_t height) -> int {
  auto retval = static_cast<int>(
      hipMemcpy2D(dest, dpitch, src, spitch, width, height, hipMemcpyDeviceToDevice));
  return retval;
}

auto memcpy_2d_inferred(
    void *dest,
    size_t dpitch,
    const void *src,
    size_t spitch,
    size_t width,
    size_t height) -> int {
  auto retval = static_cast<int>(
      hipMemcpy2D(dest, dpitch, src, spitch, width, height, hipMemcpyDefault));
  return retval;
}

} // namespace fwgpu
